#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "vec_arb_len_dot.h"

#define BLOCK_SIZE  16  // number of threads in a direction of the block

using namespace std;

__global__ void dot(float *a, float *b, float *c, int N){
    int tid = threadIdx.x + blockIdx.x*blockDim.x;  // handle the data at this index

    while(tid < N*N)
    {
        c[tid] = a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }
}

int cuda_vec_dot( float *a, float *b, float *c, int N)
{
    //int N = 9;
    //int i,j;

    //float *a, *b, *c;
    //c = (float*)malloc(N*N*sizeof(float));  
    //a = (float*)malloc(N*N*sizeof(float));
    //b = (float*)malloc(N*N*sizeof(float));

    float *dev_a, *dev_b, *dev_c; 
  
    hipMalloc((void**)&dev_c, N*N*sizeof(float));
    hipMalloc((void**)&dev_b, N*N*sizeof(float));
    hipMalloc((void**)&dev_a, N*N*sizeof(float));

    hipMemcpy(dev_a, a, N*N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N*N*sizeof(float), hipMemcpyHostToDevice);

    dot <<<(N*N+BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE>>>(dev_a, dev_b, dev_c, N);

    hipMemcpy(c, dev_c, N*N*sizeof(int), hipMemcpyDeviceToHost);
 
    hipFree(dev_c);
    hipFree(dev_b);
    hipFree(dev_a);


    return 0;
}
