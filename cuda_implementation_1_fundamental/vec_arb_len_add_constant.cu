#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "vec_arb_len_add_constant.h"

#define BLOCK_SIZE  16  // number of threads in a direction of the block

using namespace std;

__global__ void add_constant(float *a, float lambda, float *c, int N){
    int tid = threadIdx.x + blockIdx.x*blockDim.x;  // handle the data at this index

    while(tid < N*N)
    {
        c[tid] = a[tid] + lambda;
        tid += blockDim.x * gridDim.x;
    }
}

int cuda_vec_add_constant( float *a, float lambda, float *c, int N)
{
    //int N = 9;
    //int i,j;

    //float *a, *b, *c;
    //c = (float*)malloc(N*N*sizeof(float));  
    //a = (float*)malloc(N*N*sizeof(float));
    //b = (float*)malloc(N*N*sizeof(float));

    float *dev_a,  *dev_c; 
  
    hipMalloc((void**)&dev_c, N*N*sizeof(float));
    //hipMalloc((void**)&dev_b, N*N*sizeof(float));
    hipMalloc((void**)&dev_a, N*N*sizeof(float));

    hipMemcpy(dev_a, a, N*N*sizeof(float), hipMemcpyHostToDevice);
    //hipMemcpy(dev_b, b, N*N*sizeof(float), hipMemcpyHostToDevice);

    add_constant <<<(N*N+BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE>>>(dev_a, lambda, dev_c, N);

    hipMemcpy(c, dev_c, N*N*sizeof(int), hipMemcpyDeviceToHost);
 
    hipFree(dev_c);
    //hipFree(dev_b);
    hipFree(dev_a);


    return 0;
}
