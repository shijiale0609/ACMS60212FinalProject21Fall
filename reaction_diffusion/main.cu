#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

#define N 200

__global__ void discrete_laplacian(double* M, double* L){
    int tid = blockIdx.x; // handle the data at this index
    if (tid < N*N) {
        L[tid] = (-4) * M[tid];
        if (tid%N!=0) {
            L[tid] += M[tid-1];
        } else {
            L[tid] += M[tid+N-1];
        }
        if (tid%N!=N-1) {
            L[tid] += M[tid+1];
        } else {
            L[tid] += M[tid-N+1];
        }
        if (tid/N!=0) {
            L[tid] += M[tid-N];
        } else {
            L[tid] += M[tid+N*N-N];
        }
        if (tid/N!=N-1) {
            L[tid] += M[tid+N];
        } else {
            L[tid] += M[tid-N*N+N];
        }
    }
}

__global__ void diff_Matrix_A(double* dev_A, double* dev_B, double* LA, double* diff_A, double DA, double f, double k, double delta_t){
    int tid = blockIdx.x; // handle the data at this index
    if (tid < N*N) {
        diff_A[tid] = (DA*LA[tid] - dev_A[tid]*dev_B[tid]*dev_B[tid] + f*(1-dev_A[tid])) * delta_t;
    }
}

__global__ void diff_Matrix_B(double* dev_A, double* dev_B, double* LB, double* diff_B, double DB, double f, double k, double delta_t){
    int tid = blockIdx.x; // handle the data at this index
    if (tid < N*N) {
        diff_B[tid] = (DB*LB[tid] + dev_A[tid]*dev_B[tid]*dev_B[tid] - (k+f)*dev_B[tid]) * delta_t;
    }
}

__global__ void add2to1(double* M1, double* M2){
    int tid = blockIdx.x; // handle the data at this index
    if (tid < N*N) {
        M1[tid] += M2[tid];
    }
}

void gray_scott_update(double* dev_A, double* dev_B, double* LA, double* LB, double* diff_A, double* diff_B, double DA, double DB, double f, double k, double delta_t){
    discrete_laplacian<<<N*N,1>>>(dev_A, LA);
    discrete_laplacian<<<N*N,1>>>(dev_B, LB);
    diff_Matrix_A<<<N*N,1>>>(dev_A,dev_B,LA,diff_A,DA,f,k,delta_t);
    diff_Matrix_B<<<N*N,1>>>(dev_A,dev_B,LB,diff_B,DB,f,k,delta_t);
    add2to1<<<N*N,1>>>(dev_A,diff_A);
    add2to1<<<N*N,1>>>(dev_B,diff_B);
}

void get_initial_configuration(double* A0, double* B0, double random_influence){
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            int N2 = N/2, r = N/10;
            if (i < N2+r && i >= N2-r && j < N2+r && j >= N2-r) {
                A0[N*i+j] = 0.50;
                B0[N*i+j] = 0.25;
            } else {
                A0[N*i+j] = 1 - random_influence + random_influence * ((double) rand() / (RAND_MAX));
                B0[N*i+j] = random_influence * ((double) rand() / (RAND_MAX));
            }
        }
    }
}

void output_txt(double* A0, double* B0, double* A, double* B){
    FILE *fptr;
    fptr = fopen("A0.txt","w");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            fprintf(fptr, "%lf\t", A0[N*i+j]);
        }
        fprintf(fptr, "\n");
    }
    fptr = fopen("B0.txt","w");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            fprintf(fptr, "%lf\t", B0[N*i+j]);
        }
        fprintf(fptr, "\n");
    }
    fptr = fopen("A.txt","w");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            fprintf(fptr, "%lf\t", A[N*i+j]);
        }
        fprintf(fptr, "\n");
    }
    fptr = fopen("B.txt","w");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            fprintf(fptr, "%lf\t", B[N*i+j]);
        }
        fprintf(fptr, "\n");
    }
}

int main () {
    double delta_t = 1.0;
    double DA = 0.16; // second set: 0.14
    double DB = 0.08; // second set: 0.06
    double f = 0.060; // second set: 0.035
    double k = 0.062; // second set: 0.065
    int N_simulation_steps = 10000;
    double random_influence = 0.2;
    double *dev_A, *dev_B, *LA, *LB, *diff_A, *diff_B;
    double A[N*N], B[N*N], A0[N*N], B0[N*N];
    hipMalloc((void**)&dev_A, N*N*sizeof(double));
    hipMalloc((void**)&dev_B, N*N*sizeof(double));
    hipMalloc((void**)&LA, N*N*sizeof(double));
    hipMalloc((void**)&LB, N*N*sizeof(double));
    hipMalloc((void**)&diff_A, N*N*sizeof(double));
    hipMalloc((void**)&diff_B, N*N*sizeof(double));
    get_initial_configuration(A0, B0, random_influence);
    hipMemcpy(dev_A, A0, N*N*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_B, B0, N*N*sizeof(double), hipMemcpyHostToDevice);
    for (int t=0; t<N_simulation_steps; t++) {
        gray_scott_update(dev_A, dev_B, LA, LB, diff_A, diff_B, DA, DB, f, k, delta_t);
    }
    hipMemcpy(A, dev_A, N*N*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(B, dev_B, N*N*sizeof(double), hipMemcpyDeviceToHost);
    output_txt(A0, B0, A, B);
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(LA);
    hipFree(LB);
    hipFree(diff_A);
    hipFree(diff_B);
    return 0;
}
